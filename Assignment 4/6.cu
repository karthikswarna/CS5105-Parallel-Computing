#include "hip/hip_runtime.h"
%%cu
/*
 * INPUT:
 * The number of nodes and source vertex is given as input.
 * 0 <= SOURCE < NUM_NODES
 * OUTPUT:
 * If a node is not reachable from the source, 2147483647(Infinity) will be the shortest distance from s to d.
 */

#include <iostream>
#include <stdlib.h>
#include <time.h>

#define NUM_NODES 5
#define SOURCE 0

using namespace std;

#define HANDLE_ERROR( err ) ( HandleError( err, __FILE__, __LINE__ ) )

static void HandleError( hipError_t err, const char *file, int line )
{
    if (err != hipSuccess)
	{
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

__global__ void CUDA_SSSP_KERNEL1(int *Va, int *Ea, int *Wa, bool *Ma, int *Ca, int *Ua, bool *done)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid > NUM_NODES)
		*done = false;

	if (Ma[tid] == true)
	{
		Ma[tid] = false;
		
		__syncthreads();
		
		int start = Va[tid];
		int end = Va[tid + 1] - 1;
		for (int i = start; i <= end; i++) 
		{
			int nid = Ea[i];

			if(Ua[nid] > Ca[tid] + Wa[i])
			{
                Ua[nid] = Ca[tid] + Wa[i];
				*done = false;
			}
		}
	}
}

__global__ void CUDA_SSSP_KERNEL2(int *Va, int *Ea, int *Wa, bool *Ma, int *Ca, int *Ua, bool *done)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(Ca[tid] > Ua[tid])
    {
        Ca[tid] = Ua[tid];
        Ma[tid] = true;
    }
    Ua[tid] = Ca[tid];
}

int main()
{
	srand(time(NULL));

	/* Generating a random graph */
	cout << "Adjacent Matrix:" << endl;

	int adj_matrix[NUM_NODES][NUM_NODES];
	int NUM_EDGES = 0;
	for(int i = 0; i < NUM_NODES; i++)
	{
		for(int j = 0; j < NUM_NODES; j++)
		{
			adj_matrix[i][j] = rand() % 100;	// Weights can be from 0 to 100.
			if(adj_matrix[i][j] != 0)
				NUM_EDGES++;

			cout << adj_matrix[i][j] << " ";
		}
		cout << endl;
	}

	/* Creating vertex and edges array using the adjacency matrix in CPU */
	int vertices[NUM_NODES];
	int *edges = new int[NUM_EDGES];
	int *weights = new int[NUM_EDGES];
	int index = 0;
	vertices[0] = 0;
	for(int i = 0; i < NUM_NODES; i++)
	{
		// Finding the number of edges for ith vertex.
		for(int j = 0; j < NUM_NODES; j++)
		{
			if(adj_matrix[i][j] != 0)
			{
				edges[index] = j;
				weights[index] = adj_matrix[i][j];
				index++;
			}
		}

		vertices[i + 1] = index;
		if(vertices[i] == vertices[i + 1])
			vertices[i] = -1;
	}

	cout << "Vertices Array:" << endl;
	for(int i = 0; i < NUM_NODES; i++)
		cout << vertices[i] << " ";

	cout << endl << "Edges Array:" << endl;
	for(int i = 0; i < NUM_EDGES; i++)
		cout << edges[i] << " ";

	cout << endl << "Weights Array:" << endl;
	for(int i = 0; i < NUM_EDGES; i++)
		cout << weights[i] << " ";
	cout << endl;

	/* Creating and Initializing the Mask, Cost, UpdateCost arrays in CPU. */
	bool mask[NUM_NODES] = {false};
	int cost[NUM_NODES];
    int updateCost[NUM_NODES];
    for (int i = 0; i < NUM_NODES ; i++)
    {
        cost[i] = INT_MAX;
        updateCost[i] = INT_MAX;
    }

	mask[SOURCE] = true;
	cost[SOURCE] = 0;
    updateCost[SOURCE] = 0;


	/* Allocating the memory and initializing the Vertex, Edges, Frontier, Visited, Cost arrays in GPU */
	int* Va, *Ea, *Wa, *Ca, *Ua;
    bool* Ma;

	HANDLE_ERROR( hipMalloc((void**)&Va, sizeof(int)*NUM_NODES) );
	HANDLE_ERROR( hipMalloc((void**)&Ea, sizeof(int)*NUM_EDGES) );
	HANDLE_ERROR( hipMalloc((void**)&Wa, sizeof(int)*NUM_EDGES) );
	HANDLE_ERROR( hipMalloc((void**)&Ma, sizeof(bool)*NUM_NODES) );
	HANDLE_ERROR( hipMalloc((void**)&Ca, sizeof(int)*NUM_NODES) );
	HANDLE_ERROR( hipMalloc((void**)&Ua, sizeof(int)*NUM_NODES) );

	HANDLE_ERROR( hipMemcpy(Va, vertices, sizeof(int)*NUM_NODES, hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(Ea, edges, sizeof(int)*NUM_EDGES, hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(Wa, weights, sizeof(int)*NUM_EDGES, hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(Ma, mask, sizeof(bool)*NUM_NODES, hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(Ca, cost, sizeof(int)*NUM_NODES, hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(Ua, updateCost, sizeof(int)*NUM_NODES, hipMemcpyHostToDevice) );

	// int threadsPerBlock = 512;
    // int blocksPerGrid = (NUM_NODES + threadsPerBlock - 1) / threadsPerBlock;

	int blocksPerGrid = 1;
	int threadsPerBlock = NUM_NODES;

	bool done;
	bool* d_done;
	HANDLE_ERROR( hipMalloc((void**)&d_done, sizeof(bool)) );

  	done = false;
	while (!done)
    {
		done = true;
		HANDLE_ERROR( hipMemcpy(d_done, &done, sizeof(bool), hipMemcpyHostToDevice) );
		CUDA_SSSP_KERNEL1 <<<blocksPerGrid, threadsPerBlock>>>(Va, Ea, Wa, Ma, Ca, Ua, d_done);
		HANDLE_ERROR( hipMemcpy(&done, d_done , sizeof(bool), hipMemcpyDeviceToHost) );

		CUDA_SSSP_KERNEL2 <<<blocksPerGrid, threadsPerBlock>>>(Va, Ea, Wa, Ma, Ca, Ua, d_done);
	}

	HANDLE_ERROR( hipMemcpy(cost, Ca, sizeof(int)*NUM_NODES, hipMemcpyDeviceToHost) );

	cout << endl << "Shortest Path Costs from vertex " << SOURCE << ":  (2147483647 means infinity)" << endl;
	for (int i = 0; i < NUM_NODES; i++)
		cout << cost[i] << " ";
	cout << endl;

	delete[] edges;
	delete[] weights;
    hipFree (Va);
    hipFree (Ea);
    hipFree (Wa);
    hipFree (Ma);
    hipFree (Ca);
    hipFree (Ua);
    hipFree (d_done);

    return 0;
}