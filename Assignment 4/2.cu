#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>

#define HANDLE_ERROR( err ) ( HandleError( err, __FILE__, __LINE__ ) )

static void HandleError( hipError_t err, const char *file, int line )
{
    if (err != hipSuccess)
    {
        printf( "%s in %s at line %d\n", hipGetErrorString(err), file, line );
        exit(EXIT_FAILURE);
    }
}

const short N = 10;

__global__ void Vector_Addition (const int *dev_a, const int *dev_b, int *dev_c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < N) 
        dev_c [tid] = dev_a[tid] + dev_b[tid];
}

int main(void)
{
    int Host_a[N], Host_b[N], Host_c[N];
    int *dev_a , *dev_b, *dev_c ;

    HANDLE_ERROR ( hipMalloc((void **)&dev_a , N*sizeof(int) ) );
    HANDLE_ERROR ( hipMalloc((void **)&dev_b , N*sizeof(int) ) );
    HANDLE_ERROR ( hipMalloc((void **)&dev_c , N*sizeof(int) ) );

    srand(time(NULL));
    for(int i = 0; i < N; i++)
    {
        Host_a[i] = rand() % 100;
        Host_b[i] = rand() % 100; 
    }

    HANDLE_ERROR (hipMemcpy (dev_a , Host_a , N*sizeof(int) , hipMemcpyHostToDevice));
    HANDLE_ERROR (hipMemcpy (dev_b , Host_b , N*sizeof(int) , hipMemcpyHostToDevice));

    int threadsPerBlock = 512;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    Vector_Addition <<<blocksPerGrid, threadsPerBlock>>> (dev_a , dev_b , dev_c ) ;

    HANDLE_ERROR (hipMemcpy(Host_c , dev_c , N*sizeof(int) , hipMemcpyDeviceToHost));

    printf("Result array: \n");
    for (int i = 0; i < N; i++)
        printf("%d + %d = %d\n", Host_a[i], Host_b[i], Host_c[i]);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
